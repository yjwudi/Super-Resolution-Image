#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "my_image.h"

#define alpha 1.25
#define RATIO 3

using namespace cv;
using namespace std;

__global__ void find_nearest(int *r_data, int *g_data, int *b_data, size_t o_pitch, int ori_col, int *train_r, int *train_g, int *train_b, size_t d_pitch, int train_data_row, int train_data_col, int *dx, int *dy)
{
    int thread_id = threadIdx.x;
    int j = 0, s = 0, t = 0, k = 0;
    int *f_r_row, *f_g_row, *f_b_row, *s_r_row, *s_g_row, *s_b_row; 
    int *tf_r_row, *tf_g_row, *tf_b_row, *ts_r_row, *ts_g_row, *ts_b_row;
    int diff = 0, min_diff = 10000;
    if(thread_id%2 == 0)
    {
    f_r_row = (int*)((char*)r_data);
    f_g_row = (int*)((char*)g_data);
    f_b_row = (int*)((char*)b_data);
    s_r_row = (int*)((char*)r_data + o_pitch);
    s_g_row = (int*)((char*)r_data + o_pitch);
    s_b_row = (int*)((char*)r_data + o_pitch);
    k = thread_id;
        for(s = 0; s < train_data_row-2; s++)
        {
            tf_r_row = (int*)((char*)train_r + s*d_pitch);
            tf_g_row = (int*)((char*)train_g + s*d_pitch);
            tf_b_row = (int*)((char*)train_b + s*d_pitch);
            ts_r_row = (int*)((char*)train_r + (s+1)*d_pitch);
            ts_g_row = (int*)((char*)train_g + (s+1)*d_pitch);
            ts_b_row = (int*)((char*)train_b + (s+1)*d_pitch);
            for(t = 0; t < train_data_col-2; t++)
            {
                if(k+1 < ori_col)
                {
                diff = 0;
                for(j = 0; j < 2; j++)
                {
                    diff = diff + abs(f_r_row[k+j]-tf_r_row[t+j]) + abs(f_g_row[k+j]-tf_g_row[t+j]) + abs(f_b_row[k+j]-tf_b_row[t+j]) + abs(s_r_row[k+j]-ts_r_row[t+j]) + abs(s_g_row[k+j]-ts_g_row[t+j]) + abs(s_b_row[k+j]-ts_b_row[t+j]);
                }
                if(diff < min_diff)
                {
                    min_diff = diff;
                    dx[k/2] = s, dy[k/2] = t;
                }
                }
            }
        }
    }

}

//FILE *fp=fopen("output.txt", "w+");

void sr_single(Mat ori_img);
//void print_img_data(my_image img_data);
Mat data_to_img(Mat high_img, my_image img_data);
void construct(my_image ori_img_data, my_image low_img_data, double ratio, my_image high_img_data);
int main(int argc, char* argv[])
{
    system("ulimit -s unlimited");
    Mat img = imread(argv[1]);

    cout << "start: " << endl;
    sr_single(img);
    cout << "end\n";

    //fclose(fp);
    return 0;
}
void sr_single(Mat ori_img)
{
    int level = 0, total_level = 3;
    Mat up_img, down_img, tmp_high_img, tmp_low_img, high_img;
    my_image high_img_data, low_img_data, ori_img_data;
    int up_row = int(ori_img.rows*alpha+0.5), up_col = int(ori_img.cols*alpha+0.5);
    int down_row = ori_img.rows/(alpha*1.0), down_col = ori_img.cols/(alpha*1.0);
    double ratio = 1.0;
    char filename[10] = {"sri.png"};

    ori_img_data = my_image(ori_img);
    tmp_high_img = ori_img, tmp_low_img = ori_img;

    for(level = 0; level < total_level; level++)
    {
        printf("level: %d\n", level);
        printf("up_row, up_col, down_row, down_col: %d %d %d %d\n", up_row, up_col, down_row, down_col);
        resize(ori_img, up_img, Size(up_row, up_col));
        resize(ori_img, down_img, Size(down_row, down_col));

        ratio *= alpha;
        high_img_data = my_image(up_img), low_img_data = my_image(down_img);
        construct(ori_img_data, low_img_data, ratio, high_img_data);
        high_img = data_to_img(up_img, high_img_data);
        filename[2] = level+'0';
        imwrite(filename, high_img);

        up_row = int(up_row*alpha+0.5), up_col = int(up_col*alpha+0.5);
        down_row = down_row/(alpha*1.0), down_col = down_col/(alpha*1.0);
    }

}

void construct(my_image ori_img_data, my_image low_img_data, double ratio, my_image high_img_data)
{
    size_t d_pitch, o_pitch;
    int i = 0, j = 0, s = 0, t = 0, k = 0;
    int train_data_row = low_img_data.img_row, train_data_col = low_img_data.img_col;
    int row = ori_img_data.img_row, col = ori_img_data.img_col;
printf("row, col: %d %d\n", row, col);
    int *x, *y, *dx, *dy;
    int ori_r[2][col], ori_g[2][col], ori_b[2][col];
    int train_r[train_data_row][train_data_col], train_g[train_data_row][train_data_col], train_b[train_data_row][train_data_col];
    int *d_train_r, *d_train_g, *d_train_b, *d_r, *d_g, *d_b;
    int start_row, start_col, aim_row, aim_col;

    //注意rgb顺序
    for(i = 0; i < train_data_row; i++)
        for(j = 0; j < train_data_col; j++)
        {
            train_r[i][j] = low_img_data.r_data[i][j];
            train_g[i][j] = low_img_data.g_data[i][j];
            train_b[i][j] = low_img_data.b_data[i][j];

        }
    //这几个d_pitch输出看一下
    hipMallocPitch((void**)&d_train_r, &d_pitch, sizeof(int)*train_data_col, train_data_row);
    hipMallocPitch((void**)&d_train_g, &d_pitch, sizeof(int)*train_data_col, train_data_row);
    hipMallocPitch((void**)&d_train_b, &d_pitch, sizeof(int)*train_data_col, train_data_row);
    hipMemcpy2D(d_train_r, d_pitch, train_r, sizeof(int)*train_data_col, sizeof(int)*train_data_col, train_data_row, hipMemcpyHostToDevice);
    hipMemcpy2D(d_train_g, d_pitch, train_g, sizeof(int)*train_data_col, sizeof(int)*train_data_col, train_data_row, hipMemcpyHostToDevice);
    hipMemcpy2D(d_train_b, d_pitch, train_b, sizeof(int)*train_data_col, sizeof(int)*train_data_col, train_data_row, hipMemcpyHostToDevice);

    x = new int[col], y = new int[col];
    hipMalloc((void **)&dx, sizeof(int)*col);
    hipMalloc((void **)&dy, sizeof(int)*col);
    hipMallocPitch((void**)&d_r, &o_pitch, sizeof(int)*col, 2);
    hipMallocPitch((void**)&d_g, &o_pitch, sizeof(int)*col, 2);
    hipMallocPitch((void**)&d_b, &o_pitch, sizeof(int)*col, 2);
    for(i = 0; i < row-1; i+=2)
    {
        //这一段通过直接cudamemcpy原数组不借助二维数组赋值可不可以
        for(j = 0; j < col; j++)
        {
            ori_r[0][j] = ori_img_data.r_data[i][j], ori_r[1][j] = ori_img_data.r_data[i+1][j];
            ori_g[0][j] = ori_img_data.g_data[i][j], ori_g[1][j] = ori_img_data.g_data[i+1][j];
            ori_b[0][j] = ori_img_data.b_data[i][j], ori_b[1][j] = ori_img_data.b_data[i+1][j];
        }
        hipMemcpy2D(d_r, o_pitch, ori_r, sizeof(int)*col, sizeof(int)*col, 2, hipMemcpyHostToDevice);
        hipMemcpy2D(d_g, o_pitch, ori_g, sizeof(int)*col, sizeof(int)*col, 2, hipMemcpyHostToDevice);
        hipMemcpy2D(d_b, o_pitch, ori_b, sizeof(int)*col, sizeof(int)*col, 2, hipMemcpyHostToDevice);

        //dim3 blocks(1,2);
        //dim3 threads(col/2,2);
        find_nearest<<<1, col>>>(d_r, d_g, d_b, o_pitch, col, d_train_r, d_train_g, d_train_b, d_pitch, train_data_row, train_data_col, dx, dy);
        hipMemcpy(x, dx, sizeof(int)*(col/2), hipMemcpyDeviceToHost);
        hipMemcpy(y, dy, sizeof(int)*(col/2), hipMemcpyDeviceToHost);

        for(k = 0; k < col/2; k++)
        {
            aim_row = x[k]*ratio, aim_col = y[k]*ratio;
            start_row = aim_row*ratio, start_col = aim_col*ratio;
            s = (int)(2*ratio+0.5), t = (int)(2*ratio+0.5);
            if(start_row+s<high_img_data.img_row&&start_col+t<high_img_data.img_col&&aim_row+s<ori_img_data.img_row&&aim_col+t<ori_img_data.img_col)
            {//printf("k: %d col: %d\n", k, col);
              for(s = 0; s < (int)(2*ratio+0.5); s++)
                for(t = 0; t < (int)(2*ratio+0.5); t++)
                {
                    high_img_data.r_data[start_row+s][start_col+t] = ori_img_data.r_data[aim_row+s][aim_col+t];
                    high_img_data.g_data[start_row+s][start_col+t] = ori_img_data.g_data[aim_row+s][aim_col+t];
                    high_img_data.b_data[start_row+s][start_col+t] = ori_img_data.b_data[aim_row+s][aim_col+t];
                }
            }
        }

    }
    hipFree(d_train_r), hipFree(d_train_g), hipFree(d_train_b);
    hipFree(dx);
    hipFree(dy);
    delete []x;
    delete []y;

}
Mat data_to_img(Mat high_img, my_image img_data)
{
    int i = 0, j = 0;
    for(i = 0; i < img_data.img_row; i++)
    {
        for(j = 0; j < img_data.img_col; j++)
        {
            high_img.at<Vec3b>(i,j)[0] = img_data.b_data[i][j];
            high_img.at<Vec3b>(i,j)[1] = img_data.g_data[i][j];
            high_img.at<Vec3b>(i,j)[2] = img_data.r_data[i][j];
        }
    }
    return high_img;
}
/*
void print_img_data(my_image img_data)
{
    int i = 0, j = 0, row = img_data.img_row, col = img_data.img_col;
    for(i = 0; i < row; i++)
    {
        for(j = 0; j < col; j++)
            fprintf(fp, "%d ", img_data.r_data[i][j]);
        fprintf(fp, "\n");
    }
    for(i = 0; i < row; i++)
    {
        for(j = 0; j < col; j++)
            fprintf(fp, "%d ", img_data.g_data[i][j]);
        fprintf(fp, "\n");
    }
    for(i = 0; i < row; i++)
    {
        for(j = 0; j < col; j++)
            fprintf(fp, "%d ", img_data.b_data[i][j]);
        fprintf(fp, "\n");
    }
}
*/
